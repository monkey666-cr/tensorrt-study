#include "hip/hip_runtime.h"
/*
** 使用.cu是希望使用CUDA的编译器NVCC, 会自动连接cuda库

** TensorRT runtime 推理过程

** 1, 创建一个runtime对象
** 2, 反序列化生成engine: runtime ---> engine
** 3, 创建一个执行上下文ExecutionContext: engine ---> context
** 4, 填充数据
** 5, 执行推理: context ---> enqueueV2
** 6, 释放资源: delete
*/

#include <vector>
#include <fstream>
#include <cassert>
#include <iostream>

#include <NvInfer.h>
#include "hip/hip_runtime.h"

class TRTLogger : public nvinfer1::ILogger
{
    void log(Severity severity, const char *msg) noexcept override
    {
        if (severity != Severity::kINFO)
            std::cout << msg << std::endl;
    }
} gLogger;

// 加载模型
std::vector<unsigned char> loadEngineModel(const std::string &filename)
{
    std::ifstream file(filename, std::ios::binary);
    assert(file.is_open() && "load engine model failed!");

    // 移动到文件末尾
    file.seekg(0, std::ios::end);
    // 获取文件大小
    size_t size = file.tellg();

    std::vector<unsigned char> data(size);
    // 移动到文件开始
    file.seekg(0, std::ios::beg);
    // 读取文件内容到data中
    file.read((char *)data.data(), size);

    // 关闭文件
    file.close();

    return data;
}

int main(int argc, char const *argv[])
{
    // ====== 1. 创建一个runtime对象 ======
    TRTLogger logger;
    nvinfer1::IRuntime *runtime = nvinfer1::createInferRuntime(logger);

    // ====== 2. 反序列化生成engine ======
    auto engineModel = loadEngineModel("./model/mlp.engine");
    // 调用runtime的反序列化方法, 生成engine, 参数分别是: 模型数据地址, 模型大小, pluginFactory
    nvinfer1::ICudaEngine *engine = runtime->deserializeCudaEngine(engineModel.data(), engineModel.size(), nullptr);

    if (!engine)
    {
        std::cout << "deserialize engine failed!" << std::endl;

        return -1;
    }

    // ====== 3. 创建一个执行上下文 ======
    nvinfer1::IExecutionContext *context = engine->createExecutionContext();

    // ====== 4. 填充数据 ======
    // 设置stream流
    hipStream_t stream;
    hipStreamCreate(&stream);

    // 数据流转: host -> device -> inference -> host
    // 输入数据
    float *host_input_data = new float[3]{2, 4, 8};
    // 输入数据大小
    int input_data_size = 3 * sizeof(float);
    // device 输入数据
    float *device_input_data = nullptr;

    // 输出数据
    float *host_output_data = new float[2]{0, 0};
    // 输出数据大小
    int output_data_size = 2 * sizeof(float);
    // device 输出数据
    float *device_output_data = nullptr;

    // 申请device内存
    hipMalloc((void **)&device_input_data, input_data_size);
    hipMalloc((void **)&device_output_data, output_data_size);

    // host ---> device
    // 参数: 目标地址, 源地址, 数据大小, 拷贝方向
    hipMemcpyAsync(device_input_data, host_input_data, input_data_size, hipMemcpyHostToDevice, stream);

    // bindings告诉Context输入输出数据的位置
    float *bindings[] = {device_input_data, device_output_data};

    // ====== 5. 执行推理 ======
    bool success = context->enqueueV2((void **)bindings, stream, nullptr);

    // 数据从device ---> host
    hipMemcpyAsync(host_output_data, device_output_data, output_data_size, hipMemcpyDeviceToHost, stream);
    // 等待流执行完毕
    hipStreamSynchronize(stream);
    // 输出结果
    std::cout << "输出结果: " << host_output_data[0] << " " << host_output_data[1] << std::endl;

    // ====== 6. 释放资源 ======
    hipStreamDestroy(stream);
    hipFree(device_input_data);
    hipFree(device_output_data);

    delete host_input_data;
    delete host_output_data;
    delete context;
    delete engine;
    delete runtime;

    return 0;
}
